#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>
#include "matmul.cuh"

const int MAX_NUM = 100;
const int MIN_NUM = -100;

int main(int argc, char const *argv[]){
    int M = 512;
    int N = 512;
    int K = 512;

    float* A = (float*)malloc(M*K*sizeof(float));
    float* B = (float*)malloc(K*N*sizeof(float));
    float* C = (float*)malloc(M*N*sizeof(float));

    for(int i=0;i<M*K;i++){
        A[i] = (float)(rand() % (MAX_NUM - MIN_NUM + 1) + MIN_NUM);
    }
    for(int i=0;i<K*N;i++){
        B[i] = (float)(rand() % (MAX_NUM - MIN_NUM + 1) + MIN_NUM);
    }

    float* d_A, d_B, d_C;

    hipError_t error_A = hipMalloc((void**)&d_A, M*K*sizeof(float));
    CUDA_CHECK(error_A);
    hipError_t error_B = hipMalloc((void**)&d_B, K*N*sizeof(float));
    CUDA_CHECK(error_B);
    hipError_t error_C = hipMalloc((void**)&d_C, M*N*sizeof(float));
    CUDA_CHECK(error_C);

    hipError_t h2d_A = hipMemcpy(d_A, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(h2d_A);
    hipError_t h2d_B = hipMemcpy(d_B, B, K*N*sizeof(float), hipMemcpyHostToDevice);
    CUDA_CHECK(h2d_B);

    matrix_multiplication_tiled(d_A, d_B, d_C, M, N, K);

    hipError_t d2h_C = hipMemcpy(C, d_C, M*N*sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CHECK(d2h_C);
}